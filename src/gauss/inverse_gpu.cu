#include <stdio.h>	
#include <errno.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#define SWAP(x, y, z)	((z) = (x),(x) = (y),(y) = (z))

#define N 5
#define	DataType	float
#define ArraySize	(N * N * sizeof(DataType))

void printMatrix(DataType *a) {
	int i, j;

	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++)
			printf("%f\t", a[j * N + i]);
		printf("\n");
	}
	printf("\n");
}

void pivotRow(hipblasHandle_t &handle, DataType *a, DataType *a_inv, int col) {
	int pivot = -1;

	hipblasIsamax(handle,
		N - col,			// Number of elements to be searched
		a + (col * N) + col,		// Starting position
		1,				// Increment in words (NOT BYTES)
		&pivot);			// Maximum element in the col
	int row = pivot - 1 + col;			// Row number with maximum element (starts with 1)

	printf("Pivot: %d\nRow: %d\n", pivot, row);
	if(row == col)
		return;

	hipblasSswap(handle,
		N,				// Nuber of elements to be swapped
		a + col,			// Current row
		N,				// Increment (becuase of column major)
		a + row,			// Row with max pivot
		N);
	hipblasSswap(handle, N, a_inv + col, N, a_inv + row, N);
}

void normalizeRow(hipblasHandle_t &handle, DataType *a, DataType *a_inv, int row) {
	DataType scalar;

	hipMemcpy(&scalar, &a[row * N + row], sizeof(DataType), hipMemcpyDeviceToHost);
	scalar = 1 / scalar;
	hipblasSscal(handle, N, &scalar, a + row, N);
	hipblasSscal(handle, N, &scalar, a_inv + row, N);
}

__global__
void transform_matrix(DataType *a, DataType *a_inv, int row) {
	__shared__ DataType scalars[N];
	__shared__ DataType currRowA[N], currRowI[N];

	// store the scalars corresponding to the column 'row'
	scalars[threadIdx.x] = a[row * N + threadIdx.x];
	currRowA[threadIdx.x] = a[threadIdx.x * N + row];
	currRowI[threadIdx.x] = a_inv[threadIdx.x * N + row];
	__syncthreads();

	// No need to transform 'row'th row
	if(threadIdx.x == row)
		return;

	// Each thread transforms row
	for(int i = 0; i < N; i++) {
		a[i * N + threadIdx.x] -= (scalars[threadIdx.x] * currRowA[i]);
		a_inv[i * N + threadIdx.x] -= (scalars[threadIdx.x] * currRowI[i]);
	}
}

void invert(hipblasHandle_t &handle, DataType *a, DataType *a_inv) {
	for(int i = 0; i < N; i++) {
		// Pivot the matrix
		pivotRow(handle, a, a_inv, i);

		// Make column entry to be one
		normalizeRow(handle, a, a_inv, i);

		// Number of threads equals number of rows
		transform_matrix<<<1, N>>>(a, a_inv, i);
	}
}

int main(int argc, char *argv[]) {
	DataType *a, *a_inv;
	DataType *dev_a, *dev_a_inv;
	hipblasHandle_t handle;

	/* Pre-processing steps */
	if(!(a = (DataType *)malloc(ArraySize))) {
		perror("");
		return errno;
	}
	if(!(a_inv = (DataType *)malloc(ArraySize))) {
		perror("");
		return errno;
	}
	hipMalloc(&dev_a, ArraySize);
	hipMalloc(&dev_a_inv, ArraySize);
	hipblasCreate(&handle);

	/* Input column major matrix */
	for(int i = 0; i < N; i++)
		for(int j = 0; j < N; j++) {
			scanf("%f", &a[i * N + j]);
			if(i == j)
				a_inv[i * N + j] = 1;
			else
				a_inv[i * N + j] = 0;
		}
	hipMemcpy(dev_a, a, ArraySize, hipMemcpyHostToDevice);
	hipMemcpy(dev_a_inv, a_inv, ArraySize, hipMemcpyHostToDevice);

	/* Invert the matrix */
	invert(handle, dev_a, dev_a_inv);

	/* Display the result */
	hipMemcpy(a, dev_a, ArraySize, hipMemcpyDeviceToHost);
	hipMemcpy(a_inv, dev_a_inv, ArraySize, hipMemcpyDeviceToHost);
	printf("Inverse is:\n");
	//printMatrix(a);
	printMatrix(a_inv);

	/* Cleanup the mess */
	free(a);
	free(a_inv);
	hipFree(dev_a);
	hipFree(dev_a_inv);
	hipblasDestroy(handle);

	return 0;
}